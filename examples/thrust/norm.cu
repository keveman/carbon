#include "hip/hip_runtime.h"
/*
 *  Copyright 2010-2012 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cmath>

#include <carbon/lambda/primitives.hpp>
#include <carbon/lambda/operators.hpp>

//   This example computes the norm [1] of a vector.  The norm is 
// computed by squaring all numbers in the vector, summing the 
// squares, and taking the square root of the sum of squares.  In
// Thrust this operation is efficiently implemented with the 
// transform_reduce() algorith.  Specifically, we first transform
// x -> x^2 and the compute a standard plus reduction.  Since there
// is no built-in functor for squaring numbers, we define our own
// square functor.
//
// [1] http://en.wikipedia.org/wiki/Norm_(mathematics)#Euclidean_norm


using namespace carbon::lambda;
int main(void)
{
    // initialize host array
    float x[4] = {1.0, 2.0, 3.0, 4.0};

    // transfer to device
    thrust::device_vector<float> d_x(x, x + 4);

    float init = 0;

    // compute norm
    float norm = std::sqrt( thrust::transform_reduce(d_x.begin(), d_x.end(),
                                                     _1*_1,
                                                     init,
                                                     _1+_2) );

    std::cout << "norm is " << norm << std::endl;

    return 0;
}

