#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/extrema.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>

#include <iomanip>
#include <stdio.h>
#include <cmath>
#include <carbon/lambda.hpp>

using namespace carbon::lambda;

// Compute an approximate Voronoi Diagram with a Jump Flooding Algorithm (JFA)
//
// References
//   http://en.wikipedia.org/wiki/Voronoi_diagram
//   http://www.comp.nus.edu.sg/~tants/jfa.html
//   http://www.utdallas.edu/~guodongrong/Papers/Dissertation.pdf
//
// Thanks to David Coeurjolly for contributing this example



// minFunctor
// Tuple  = <seeds,seeds + k,seeds + m*k, seeds - k, 
//           seeds - m*k, seeds+ k+m*k,seeds + k-m*k,
//           seeds- k+m*k,seeds - k+m*k, i>

struct minVoro_impl {
  template<typename, typename, typename, typename, typename, typename>
  struct result { typedef int type; };

  __host__ __device__
  int operator()(int x_i, int y_i, int p, int q, int m, int n) const
  {    
    if (q == m*n)
      return p;

    // coordinates of points p and q
    int y_q =  q / m;
    int x_q =  q - y_q * m;
    int y_p =  p / m;
    int x_p =  p - y_p * m;
        
    // squared distances
    int d_iq = (x_i-x_q) * (x_i-x_q) + (y_i-y_q) * (y_i-y_q);
    int d_ip = (x_i-x_p) * (x_i-x_p) + (y_i-y_p) * (y_i-y_p);

    if (d_iq < d_ip)
      return q;  // q is closer
    else
      return p;
  }
};

function<minVoro_impl> const minVoro_;

struct minFunctor
{
  int k,m,n;
  
  __host__ __device__
    minFunctor(int _m,int _n,int _k) : n(_n),m(_m),k(_k) {}


  //To decide I have to change my current Voronoi site
  __host__ __device__
      int minVoro(int x_i, int y_i, int p, int q)
      {    
          if (q == m*n)
              return p;

          // coordinates of points p and q
          int y_q =  q / m;
          int x_q =  q - y_q * m;
          int y_p =  p / m;
          int x_p =  p - y_p * m;
        
          // squared distances
          int d_iq = (x_i-x_q) * (x_i-x_q) + (y_i-y_q) * (y_i-y_q);
          int d_ip = (x_i-x_p) * (x_i-x_p) + (y_i-y_p) * (y_i-y_p);

          if (d_iq < d_ip)
              return q;  // q is closer
          else
              return p;
      }

  //For each point p+{-k,0,k}, we keep the Site with minimum distance
  template <typename Tuple>
  __host__ __device__
  int operator()(const Tuple &t)
  {
      //Current point and site
      int i = thrust::get<9>(t);
      int v = thrust::get<0>(t);

      //Current point coordinates
      int y = i / m;    
      int x = i - y * m;

      if (x >= k)
      {
          v = minVoro(x, y, v, thrust::get<3>(t));

          if (y >= k)
              v = minVoro(x, y, v, thrust::get<8>(t));

          if (y + k < n)
              v = minVoro(x, y, v, thrust::get<7>(t));
      }

      if (x + k < m)
      { 
          v = minVoro(x, y, v, thrust::get<1>(t));

          if (y >= k)
              v = minVoro(x, y, v, thrust::get<6>(t));
          if (y + k < n)
              v = minVoro(x, y, v, thrust::get<5>(t));
      }

      if (y >= k)
          v = minVoro(x, y, v, thrust::get<4>(t));
      if (y + k < n)
          v = minVoro(x, y, v, thrust::get<2>(t));

      //global return
      return v;
  }
};



// print an M-by-N array
template <typename T>
void print(int m, int n, const thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(int i = 0; i < m; i++)
    {
        for(int j = 0; j < n; j++)
            std::cout << std::setw(4) << h_data[i * n + j] << " ";
        std::cout << "\n";
    }
}



/********************** TIMER Functions *************/
//Global vars
hipEvent_t start;
hipEvent_t end;
void timer_start()
{
    hipEventCreate(&start); 
    hipEventCreate(&end);
    hipEventRecord(start,0);
}

float timer_stop_and_display()
{
  float elapsed_time;
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time, start, end);

  std::cout << "  ( "<< elapsed_time << "ms )" << std::endl;

  return elapsed_time;
}
/********************** TIMER Functions *************/



void generate_random_sites(thrust::host_vector<int> &t, int Nb, int m, int n)
{
  srand(m*n);

  for(int k = 0; k < Nb; k++)
  {
      int index = (m * n) * (rand() / (RAND_MAX + 1.0f));

      t[index] = index + 1;
  }
}

//Export the tab to PGM image format
void vector_to_pgm(thrust::host_vector<int> &t, int m, int n,char *out)
{
    FILE *f;

    f=fopen(out,"w+t");
    fprintf(f,"P2\n");
    fprintf(f,"%d %d\n 253\n",m,n);

    for(int j = 0; j < n ; j++)
    {
        for(int i = 0; i < m ; i++)
        {
            fprintf(f,"%d ",(int)(71*t[j*m+i])%253); //Hash function to map values to [0,255]
        }
    }
    fprintf(f,"\n");
    fclose(f);
}

/************Main Jfa loop********************/
// Perform a jump with step k
void jfa(thrust::device_vector<int>& in,thrust::device_vector<int>& out, unsigned int k, int m, int n)
{
   thrust::transform(
        thrust::make_zip_iterator(
            thrust::make_tuple(in.begin(), 
                               in.begin() + k, 
                               in.begin() + m*k, 
                               in.begin() - k, 
                               in.begin() - m*k, 
                               in.begin() + k+m*k,
                               in.begin() + k-m*k,
                               in.begin() - k+m*k,
                               in.begin() - k-m*k,
                               thrust::counting_iterator<int>(0))),
        thrust::make_zip_iterator(
            thrust::make_tuple(in.begin(), 
				    		   in.begin() + k, 
                               in.begin() + m*k, 
                               in.begin() - k, 
                               in.begin() - m*k, 
                               in.begin() + k+m*k,
                               in.begin() + k-m*k,
                               in.begin() - k+m*k,
                               in.begin() - k-m*k,
                               thrust::counting_iterator<int>(0)))+ n*m,
        out.begin(),
     let_(_v = get_<0>(_1)) [
       let_(_i = get_<9>(_1)) [
         let_(_y = _i/m) [
           let_(_x = _i - _y*m) [
             if_(_x >= k) [
               _v = minVoro_(_x, _y, _v, get_<3>(_1), m, n),
               if_(_y >= k) [
                 _v = minVoro_(_x, _y, _v, get_<8>(_1), m, n)
               ],
               if_(_y+k <= n) [
                 _v = minVoro_(_x, _y, _v, get_<7>(_1), m, n)
               ]
             ],
             if_(_x + k < m) [
                 _v = minVoro_(_x, _y, _v, get_<1>(_1), m, n),
               if_(_y >= k) [
                 _v = minVoro_(_x, _y, _v, get_<6>(_1), m, n)
               ],
               if_(_y+k < n) [
                 _v = minVoro_(_x, _y, _v, get_<5>(_1), m, n)
               ]
             ],
             if_(_y >= k) [
               _v = minVoro_(_x, _y, _v, get_<4>(_1), m, n)
             ],
             if_(_y+k < n) [
               _v = minVoro_(_x, _y, _v, get_<2>(_1), m, n)
             ]
           ]
         ]
       ]
     ]
     ); 
}
/********************************************/



int main(void)
{
  int m = 2048; // number of rows
  int n = 2048; // number of columns  
  int s = 1000; // number of sites
 
  //Host vector to encode a 2D image
  std::cout << "[Inititialize " << m << "x" << n << " Image]" << std::endl;
  timer_start();
  thrust::host_vector<int> seeds_host(m*n, m*n);
  generate_random_sites(seeds_host,s,m,n);
  timer_stop_and_display();

  std::cout<<"[Copy to Device]" << std::endl;
  timer_start();
  thrust::device_vector<int> seeds = seeds_host;
  thrust::device_vector<int> temp(seeds);
  timer_stop_and_display();

  //JFA+1  : before entering the log(n) loop, we perform a jump with k=1
  timer_start();
  std::cout<<"[JFA stepping]" << std::endl;
  jfa(seeds,temp,1,m,n);
  seeds.swap(temp);
 
  //JFA : main loop with k=n/2, n/4, ..., 1
  for(int k = thrust::max(m,n) / 2; k > 0; k /= 2)
  {
      jfa(seeds,temp,k,m,n);
      seeds.swap(temp);
  }
  float time = timer_stop_and_display();  
  std::cout <<"  ( " <<  seeds.size() / (1e3 * time) << " MPixel/s ) " << std::endl;
  
  std::cout << "[Device to Host Copy]" << std::endl;
  timer_start();
  seeds_host = seeds;
  timer_stop_and_display();
  
  std::cout << "[PGM Export]" << std::endl;
  timer_start();
  vector_to_pgm(seeds_host, m, n, "discrete_voronoi.pgm");
  timer_stop_and_display();

  return 0;
}

