#include "hip/hip_runtime.h"
/*
 *  Copyright 2010-2012 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <iostream>
#include <carbon/lambda/primitives.hpp>
#include <carbon/lambda/operators.hpp>
#include <carbon/lambda/statements.hpp>

using namespace std;
using namespace carbon::lambda;

template<typename F>
__global__ void kernel(int *a, F f)
{
  int i=4;
  *a = 0;
  f(*a, i);
}

int main()
{
  cout << arg1(42) << "\n";
  int *da, ha;
  hipMalloc((void **)&da, sizeof(int));

  kernel<<<1,1>>>(da, while_(arg2)[arg2 = arg2-1, arg1 = arg1 + 1]);

  hipMemcpy(&ha, da, sizeof(int), hipMemcpyDeviceToHost);

  cout << ha << "\n";
  return 0;
}
