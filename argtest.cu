#include "hip/hip_runtime.h"
#include <iostream>
#include <carbon/lambda/primitives.hpp>

using namespace std;
using namespace carbon::lambda;

template<typename F>
__global__ void kernel(int *a, F f)
{
  int i=42;
  *a = f(42);
}

int main()
{
  cout << arg1(42) << "\n";
  int *da, ha;
  hipMalloc((void **)&da, sizeof(int));

  kernel<<<1,1>>>(da, arg1);

  hipMemcpy(&ha, da, sizeof(int), hipMemcpyDeviceToHost);

  cout << ha << "\n";
  return 0;
}
